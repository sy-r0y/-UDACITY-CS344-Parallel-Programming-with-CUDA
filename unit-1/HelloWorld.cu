#include "hip/hip_runtime.h"
/* 
 * Program to square elements of an array by spinning up seperate threads, each of which 
 * individually calculate the square of an element.
 *
 * .cu extension =>> CUDA extension of a normal C program.
 * 'nvcc' compiler is used to compile this program.
*/

#include<stdio.h>

__global__ void square(float *d_out,float *d_in) {

   	   int idx=threadIdx.x;
	   float f=d_in[idx]; /* the d_ of d_in is a convention of prefixing d_ for any device(GPU) 
                               * variable.
			       */
	   d_out[idx]=f*f;

} //Kernel routine ends

int main(int argc,char **argv) {
    	     
           const int ARRAY_SIZE=64;
	   const int ARRAY_BYTES=ARRAY_SIZE*sizeof(float);

	   //Generate the input array on the "Host"(i.e the CPU)    
	   float h_in[ARRAY_SIZE]; /* the h_ of h_in is a convention of prefixing h_ for any host                                         * variable.
	   	 		    */	   
           for(int i=0;i<ARRAY_SIZE;i++) {
	     h_in[i]=float(i); //Typecast the value i into a float.
	   }

	   float h_out[ARRAY_SIZE];
	   
	   //Declare GPU pointers.
	   float *d_in;
	   float *d_out;
	   
	   //Allocate GPU memory.
	   hipMalloc((void **)&d_in,ARRAY_BYTES); /* hipMalloc is similar to regular malloc, but
	                                            * it allocates memory on the GPU.
						    */
	   hipMalloc((void **)&d_out,ARRAY_BYTES);
	   
	   //Transfer array to GPU.
	   hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice); /* Memory transfer 
	   							      * direction: CPU -> GPU
								      */

           //launch kernel.
	   square<<<1,ARRAY_SIZE>>>(d_out,d_int); /* <<< >>> is the CUDA launch operator.
	   
	   //Copy results back to CPU. Transer direction: GPU -> CPU
	   hipMemcpy(h_out,d_out,ARRAY_BYTES,cudaMemcpuDeviceToHost);

	   //print resulting array
	   for(int i=0;i<ARRAY_SIZE;i++) {
	     printf("%f",h_out[i]);
	     printf(((i%4)!=3)?"\t":"\n");
	   }

	   //Free GPU memory allocation.
	   hipFree(d_in); hipFree(d_out);	   
	   return 0;
